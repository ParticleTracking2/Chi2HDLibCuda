#include "hip/hip_runtime.h"
//============================================================================
// Name        : Chi2HD_Cuda.cu
// Author      : Juan Silva
// Version     :
// Copyright   : All rights reserved
// Description : Funciones para trabajar el algoritmo de minimos cuadrados
//============================================================================

#include "Headers/Chi2HD_Cuda.h"
#include "hipfft/hipfft.h"

/**
 * Elementos a considerar con 275 GTX
 * Global Memory = 896 MB
 * Const Memory = 65KB (16000 Floats)
 * Shared Memory = 16KB (Compartida en el bloque = 4000 Floats)
 * Registros por Bloque = 16K
 * ------------------------------
 * Cuda Cores = 240
 * Max Threads x Bloque = 512
 * Maximas dimensiones por bloque = 512 x 512 x 64
 * Maximas dimensiones de una grilla = 65535 x 65535 x 1
 */

#if defined(__cplusplus)
extern "C" {
#endif

/**
 * Maximos elementos para le GPU GTX590
 */
const unsigned int STD_THREAD_SIZE = 512;
const unsigned int MAX_THREAD_SIZE = 1024;

const unsigned int STD_BLOCK_SIZE = 512;
const unsigned int MAX_BLOCK_SIZE = 1024;

const unsigned int STD_GRID_SIZE = 1024;
const unsigned int MAX_GRID_SIZE = 65535;

unsigned int _findOptimalGridSize(cuMyArray2D *arr){
	return STD_GRID_SIZE;
}
unsigned int _findOptimalBlockSize(cuMyArray2D *arr){
	return STD_BLOCK_SIZE;
}

cuMyArray2D CHI2HD_createArray(unsigned int sx, unsigned int sy){
	cuMyArray2D ret;
	hipError_t err = hipMallocPitch((void**)&ret._device_array, &ret._device_pitch, (size_t)(sx*sizeof(float)), (size_t)(sy));

	if(err != hipSuccess)
		exit(1);

	ret._host_array = 0;
	ret._device = 0;
	ret._sizeX = sx;
	ret._sizeY = sy;

	return ret;
}

bool CHI2HD_destroyArray(cuMyArray2D *arr){
	if(arr->_device_array){
		hipError_t err = hipFree(arr->_device_array);
		if(err != hipSuccess)
			exit(-1);
		return true;
	}
	free(arr->_host_array);

	arr->_device_array = 0;
	arr->_host_array = 0;
	arr->_device = -1;
	arr->_sizeX = 0;
	arr->_sizeY = 0;
	return false;
}

/******************
 * RESET
 ******************/
__global__ void __CHI2HD_reset(float* arr, int size, float def){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
		arr[idx] = def;
}

void CHI2HD_reset(cuMyArray2D *arr, float def){
	dim3 dimGrid(_findOptimalGridSize(arr));
	dim3 dimBlock(_findOptimalBlockSize(arr));
	__CHI2HD_reset<<<dimGrid, dimBlock>>>(arr->_device_array, arr->getSize(), def);
	hipDeviceSynchronize();
}

/******************
 * SQUARE
 ******************/
__global__ void __CHI2HD_squareIt(float* arr, unsigned int size){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
		arr[idx] = arr[idx] * arr[idx];
}

void CHI2HD_squareIt(cuMyArray2D *arr){
	dim3 dimGrid(_findOptimalGridSize(arr));
	dim3 dimBlock(_findOptimalBlockSize(arr));
	__CHI2HD_squareIt<<<dimGrid, dimBlock>>>(arr->_device_array, arr->getSize());
	hipDeviceSynchronize();
}

/******************
 * CUBE
 ******************/
__global__ void __CHI2HD_cubeIt(float* arr, int size){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
		arr[idx] = arr[idx] * arr[idx] * arr[idx];
}

void CHI2HD_cubeIt(cuMyArray2D *arr){
	dim3 dimGrid(_findOptimalGridSize(arr));
	dim3 dimBlock(_findOptimalBlockSize(arr));
	__CHI2HD_cubeIt<<<dimGrid, dimBlock>>>(arr->_device_array, arr->getSize());
	hipDeviceSynchronize();
}

/******************
 * Copy
 ******************/
void CHI2HD_copyToHost(cuMyArray2D *arr){
	size_t size = arr->getSize()*sizeof(float);
	if(!arr->_host_array)
		arr->_host_array = (float*)malloc(size);
	hipError_t err = hipMemcpy(arr->_host_array, arr->_device_array, size, hipMemcpyDeviceToHost);
	if(err != hipSuccess)
		exit(-1);
}

void CHI2HD_copyToDevice(cuMyArray2D *arr){
	size_t size = arr->getSize()*sizeof(float);
	hipError_t err;
	if(!arr->_device_array){
		err = hipMallocPitch((void**)&arr->_device_array, &arr->_device_pitch, arr->_sizeX*sizeof(float), arr->_sizeY);
		if(err != hipSuccess) exit(-1);
	}
	err = hipMemcpy(arr->_device_array, arr->_host_array, size, hipMemcpyHostToDevice);
	if(err != hipSuccess) exit(-1);
}

/******************
 * Min Max
 ******************/
myPair CHI2HD_minMax(cuMyArray2D *arr){
	myPair ret;
	if(!arr->_host_array)
		CHI2HD_copyToHost(arr);

	float tempMax = arr->getValueHost(0);
	float tempMin = arr->getValueHost(0);
	for(unsigned int i=0; i < arr->getSize(); ++i){
		float tmp = arr->_host_array[i];
		if(tempMax < tmp)
			tempMax = tmp;
		if(tempMin > tmp)
			tempMin = tmp;
	}
	ret.first = tempMin;
	ret.second = tempMax;

	return ret;
}

/******************
 * Normalizar
 ******************/
__global__ void __CHI2HD_normalize(float* arr, unsigned int size, float _min, float _max){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float dif = _max - _min;
	if(idx < size)
		arr[idx] = (float)((1.0f*_max - arr[idx]*1.0f)/dif);
}

void CHI2HD_normalize(cuMyArray2D *arr, float _min, float _max){
	dim3 dimGrid(_findOptimalGridSize(arr));
	dim3 dimBlock(_findOptimalBlockSize(arr));
	__CHI2HD_normalize<<<dimGrid, dimBlock>>>(arr->_device_array, arr->getSize(), _min, _max);
	hipDeviceSynchronize();
}

/******************
 * Kernel
 ******************/
__global__ void __CHI2HD_gen_kernel(float* arr, unsigned int size, unsigned int ss, unsigned int os, float d, float w){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size){
		float absolute = abs(sqrtf( (idx%ss-os)*(idx%ss-os) + (idx/ss-os)*(idx/ss-os) ));
		arr[idx] = (1.0f - tanhf((absolute - d/2.0f)/w))/2.0f;
	}
}

cuMyArray2D CHI2HD_gen_kernel(unsigned int ss, unsigned int os, float d, float w){
	cuMyArray2D kernel = CHI2HD_createArray(ss,ss);
	dim3 dimGrid(1);
	dim3 dimBlock(ss*ss);
	__CHI2HD_gen_kernel<<<dimGrid, dimBlock>>>(kernel._device_array, kernel.getSize(), ss, os, d, w);
	hipDeviceSynchronize();
	CHI2HD_copyToHost(&kernel);
	return kernel;
}

/******************
 * Convolucion 2D
 ******************/
cuMyArray2D CHI2HD_conv2D(cuMyArray2D* img, cuMyArray2D* kernel_img){
	hipfftHandle plan_forward_image, plan_forward_kernel, plan_backward;
	hipfftComplex *fft_image, *fft_kernel;
	hipfftReal *ifft_result, *data, *kernel; // float *
	size_t ifft_result_pitch, data_pitch, kernel_pitch;

	int nwidth 	=	(int)(img->_sizeX+kernel_img->_sizeX-1);
	int nheight	=	(int)(img->_sizeY+kernel_img->_sizeY-1);
	// Input Complex Data
	hipMalloc((void**)&fft_image, sizeof(hipfftComplex)*(nwidth*(floor(nheight/2) + 1)));
	hipMalloc((void**)&fft_kernel, sizeof(hipfftComplex)*(nwidth*(floor(nheight/2) + 1)));
	// Output Real Data
	hipMallocPitch((void**)&ifft_result, &ifft_result_pitch, sizeof(hipfftReal)*nwidth, nheight);
	hipMallocPitch((void**)&data, &data_pitch, sizeof(hipfftReal)*nwidth, nheight);
	hipMallocPitch((void**)&kernel, &kernel_pitch, sizeof(hipfftReal)*nwidth, nheight);

	// Plans
	hipfftPlan2d(&plan_forward_image, nwidth, nheight, HIPFFT_R2C);
	hipfftPlan2d(&plan_forward_kernel, nwidth, nheight, HIPFFT_R2C);
	hipfftPlan2d(&plan_backward, nwidth, nheight, HIPFFT_C2R);

	// Populate Data
	hipMemset((void*)data, 0, nwidth*nheight*sizeof(float));
	hipMemset((void*)kernel, 0, nwidth*nheight*sizeof(float));
	hipMemcpy2D(data, data_pitch, img->_device_array, img->_device_pitch, img->_sizeX*sizeof(float), img->_sizeY, hipMemcpyDeviceToDevice);
	hipMemcpy2D(kernel, kernel_pitch, kernel_img->_device_array, kernel_img->_device_pitch, kernel_img->_sizeX*sizeof(float), kernel_img->_sizeY, hipMemcpyDeviceToDevice);

	// Execute Plan
	hipfftExecR2C(plan_forward_image, data, fft_image);

	hipfftExecR2C(plan_forward_kernel, kernel, fft_kernel);

	// Populate final data
	// TODO

	// Execute Plan
	hipfftExecC2R(plan_backward, fft_image, ifft_result);

	// Copy Result to output;
	// TODO
	cuMyArray2D ret;

	hipfftDestroy(plan_forward_image);
	hipfftDestroy(plan_forward_kernel);
	hipfftDestroy(plan_backward);
	hipFree(data);
	hipFree(kernel);
	hipFree(ifft_result);
	hipFree(fft_image);
	hipFree(fft_kernel);

	return ret;
}

#if defined(__cplusplus)
}
#endif
