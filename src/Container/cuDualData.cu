/*
 * cuDualData.cu
 *
 *  Created on: 12/12/2011
 *      Author: juanin
 */

#include "../Headers/Container/cuDualData.h"
#include "../Headers/Chi2LibcuUtils.h"

DualDatai DualData_CreateInt(){
	DualDatai pair;

	pair.h_data = (int*)malloc(sizeof(int));
	hipError_t err = hipMalloc((void**)&pair.d_data, sizeof(int));
	manageError(err);

	pair.h_data[0] = 0;
	hipMemset(pair.d_data, 0, sizeof(int));

	return pair;
}

DualDataf DualData_CreateFloat(){
	DualDataf pair;

	pair.h_data = (float*)malloc(sizeof(float));
	hipError_t err = hipMalloc((void**)&pair.d_data, sizeof(float));
	manageError(err);

	pair.h_data[0] = 0;
	hipMemset(pair.d_data, 0, sizeof(float));

	return pair;
}

void DualData_CopyToHost(DualDatai data){
	hipError_t err = hipMemcpy(data.h_data, data.d_data, sizeof(int), hipMemcpyDeviceToHost);
	manageError(err);
}

void DualData_CopyToDevice(DualDatai data){
	hipError_t err = hipMemcpy(data.d_data, data.h_data, sizeof(int), hipMemcpyHostToDevice);
	manageError(err);
}

void DualData_CopyToHost(DualDataf data){
	hipError_t err = hipMemcpy(data.h_data, data.d_data, sizeof(float), hipMemcpyDeviceToHost);
	manageError(err);
}

void DualData_CopyToDevice(DualDataf data){
	hipError_t err = hipMemcpy(data.d_data, data.h_data, sizeof(float), hipMemcpyHostToDevice);
	manageError(err);
}

void DualData_Destroy(DualDatai data){
	free(data.h_data);
	hipFree(data.d_data);
}

void DualData_Destroy(DualDataf data){
	free(data.h_data);
	hipFree(data.d_data);
}


