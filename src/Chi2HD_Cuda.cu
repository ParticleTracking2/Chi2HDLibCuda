#include "hip/hip_runtime.h"
//============================================================================
// Name        : Chi2HD_Cuda.cu
// Author      : Juan Silva
// Version     :
// Copyright   : All rights reserved
// Description : Funciones para trabajar el algoritmo de minimos cuadrados
//============================================================================

#include "Headers/Chi2HD_Cuda.h"

/**
 * Elementos a considerar con 275 GTX
 * Global Memory = 896 MB
 * Const Memory = 65KB (16000 Floats)
 * Shared Memory = 16KB (Compartida en el bloque = 4000 Floats)
 * Registros por Bloque = 16K
 * ------------------------------
 * Cuda Cores = 240
 * Max Threads x Bloque = 512
 * Maximas dimensiones por bloque = 512 x 512 x 64
 * Maximas dimensiones de una grilla = 65535 x 65535 x 1
 */

#if defined(__cplusplus)
extern "C" {
#endif

/**
 * Maximos elementos para le GPU GTX590
 */
const unsigned int STD_THREAD_SIZE = 512;
const unsigned int MAX_THREAD_SIZE = 1024;

const unsigned int STD_BLOCK_SIZE = 512;
const unsigned int MAX_BLOCK_SIZE = 1024;

const unsigned int STD_GRID_SIZE = 1024;
const unsigned int MAX_GRID_SIZE = 65535;

unsigned int _findOptimalGridSize(cuMyArray2D *arr){
	return STD_GRID_SIZE;
}
unsigned int _findOptimalBlockSize(cuMyArray2D *arr){
	return STD_BLOCK_SIZE;
}

cuMyArray2D CHI2HD_createArray(unsigned int sx, unsigned int sy){
	cuMyArray2D ret;
	hipError_t err = hipMalloc((void**)&ret._device_array, (size_t) sx*sy*sizeof(float));

	if(err != hipSuccess)
		exit(1);

	ret._host_array = 0;
	ret._device = 0;
	ret._sizeX = sx;
	ret._sizeY = sy;

	return ret;
}

bool CHI2HD_destroyArray(cuMyArray2D *arr){
	if(arr->_device_array){
		hipError_t err = hipFree(arr->_device_array);
		if(err != hipSuccess)
			exit(-1);
		return true;
	}
	free(arr->_host_array);

	arr->_device_array = 0;
	arr->_host_array = 0;
	arr->_device = -1;
	arr->_sizeX = 0;
	arr->_sizeY = 0;
	return false;
}

/******************
 * RESET
 ******************/
__global__ void __CHI2HD_reset(float* arr, int size, float def){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
		arr[idx] = def;
}

void CHI2HD_reset(cuMyArray2D *arr, float def){
	dim3 dimGrid(_findOptimalGridSize(arr));
	dim3 dimBlock(_findOptimalBlockSize(arr));
	__CHI2HD_reset<<<dimGrid, dimBlock>>>(arr->_device_array, arr->getSize(), def);
	hipDeviceSynchronize();
}

/******************
 * SQUARE
 ******************/
__global__ void __CHI2HD_squareIt(float* arr, unsigned int size){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
		arr[idx] = arr[idx] * arr[idx];
}

void CHI2HD_squareIt(cuMyArray2D *arr){
	dim3 dimGrid(_findOptimalGridSize(arr));
	dim3 dimBlock(_findOptimalBlockSize(arr));
	__CHI2HD_squareIt<<<dimGrid, dimBlock>>>(arr->_device_array, arr->getSize());
	hipDeviceSynchronize();
}

/******************
 * CUBE
 ******************/
__global__ void __CHI2HD_cubeIt(float* arr, int size){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
		arr[idx] = arr[idx] * arr[idx] * arr[idx];
}

void CHI2HD_cubeIt(cuMyArray2D *arr){
	dim3 dimGrid(_findOptimalGridSize(arr));
	dim3 dimBlock(_findOptimalBlockSize(arr));
	__CHI2HD_cubeIt<<<dimGrid, dimBlock>>>(arr->_device_array, arr->getSize());
	hipDeviceSynchronize();
}

/******************
 * Copy
 ******************/
void CHI2HD_copyToHost(cuMyArray2D *arr){
	size_t size = arr->getSize()*sizeof(float);
	if(!arr->_host_array)
		arr->_host_array = (float*)malloc(size);
	hipError_t err = hipMemcpy(arr->_host_array, arr->_device_array, size, hipMemcpyDeviceToHost);
	if(err != hipSuccess)
		exit(-1);
}

void CHI2HD_copyToDevice(cuMyArray2D *arr){
	size_t size = arr->getSize()*sizeof(float);
	hipError_t err;
	if(!arr->_device_array){
		err = hipMalloc((void**)&arr->_device_array, size);
		if(err != hipSuccess) exit(-1);
	}
	err = hipMemcpy(arr->_device_array, arr->_host_array, size, hipMemcpyHostToDevice);
	if(err != hipSuccess) exit(-1);
}

/******************
 * Min Max
 ******************/
myPair CHI2HD_minMax(cuMyArray2D *arr){
	myPair ret;
	if(!arr->_host_array)
		CHI2HD_copyToHost(arr);

	float tempMax = arr->getValueHost(0);
	float tempMin = arr->getValueHost(0);
	for(unsigned int i=0; i < arr->getSize(); ++i){
		float tmp = arr->_host_array[i];
		if(tempMax < tmp)
			tempMax = tmp;
		if(tempMin > tmp)
			tempMin = tmp;
	}
	ret.first = tempMin;
	ret.second = tempMax;

	return ret;
}

/******************
 * Normalizar
 ******************/
__global__ void __CHI2HD_normalize(float* arr, unsigned int size, float _min, float _max){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float dif = _max - _min;
	if(idx < size)
		arr[idx] = (float)((1.0f*_max - arr[idx]*1.0f)/dif);
}

void CHI2HD_normalize(cuMyArray2D *arr, float _min, float _max){
	dim3 dimGrid(_findOptimalGridSize(arr));
	dim3 dimBlock(_findOptimalBlockSize(arr));
	__CHI2HD_normalize<<<dimGrid, dimBlock>>>(arr->_device_array, arr->getSize(), _min, _max);
	hipDeviceSynchronize();
}

/******************
 * Kernel
 ******************/
__global__ void __CHI2HD_gen_kernel(float* arr, unsigned int size, unsigned int ss, unsigned int os, float d, float w){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size){
		float absolute = abs(sqrtf( (idx%ss-os)*(idx%ss-os) + (idx/ss-os)*(idx/ss-os) ));
		arr[idx] = (1.0f - tanhf((absolute - d/2.0f)/w))/2.0f;
	}
}

cuMyArray2D CHI2HD_gen_kernel(unsigned int ss, unsigned int os, float d, float w){
	cuMyArray2D kernel = CHI2HD_createArray(ss,ss);
	dim3 dimGrid(1);
	dim3 dimBlock(ss*ss);
	__CHI2HD_gen_kernel<<<dimGrid, dimBlock>>>(kernel._device_array, kernel.getSize(), ss, os, d, w);
	hipDeviceSynchronize();
	CHI2HD_copyToHost(&kernel);
	return kernel;
}

#if defined(__cplusplus)
}
#endif
