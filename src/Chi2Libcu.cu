#include "hip/hip_runtime.h"
/*
 * Chi2Libcu.cu
 *
 *  Created on: 10/12/2011
 *      Author: juanin
 */
#include <thrust/sort.h>
#include <thrust/functional.h>
#include "Headers/Container/cuDualData.h"
#include "Headers/Chi2Libcu.h"
#include "Headers/Chi2LibcuUtils.h"

/******************
 * Min Max
 ******************/
pair<float, float> Chi2Libcu::minMax(cuMyMatrix *arr){
	pair<float, float> ret;
	arr->copyToHost();

	float tempMax = arr->getValueHost(0);
	float tempMin = arr->getValueHost(0);
	for(unsigned int i=0; i < arr->size(); ++i){
		float tmp = arr->getValueHost(i);
		if(tempMax < tmp)
			tempMax = tmp;
		if(tempMin > tmp)
			tempMin = tmp;
	}
	ret.first = tempMin;
	ret.second = tempMax;

	return ret;
}
/******************
 * Normalizar
 ******************/
__global__ void __normalize(float* arr, unsigned int size, float _min, float _max){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float dif = _max - _min;
	if(idx < size)
		arr[idx] = (float)((1.0f*_max - arr[idx]*1.0f)/dif);
}

void Chi2Libcu::normalize(cuMyMatrix *arr, float _min, float _max){
	dim3 dimGrid(_findOptimalGridSize(arr->size()));
	dim3 dimBlock(_findOptimalBlockSize(arr->size()));
	__normalize<<<dimGrid, dimBlock>>>(arr->devicePointer(), arr->size(), _min, _max);
	hipError_t err = hipDeviceSynchronize();
	manageError(err);
}


/******************
 * Kernel
 ******************/
__global__ void __gen_kernel(float* arr, unsigned int size, unsigned int ss, unsigned int os, float d, float w){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size){
		float absolute = abs(sqrtf( (idx%ss-os)*(idx%ss-os) + (idx/ss-os)*(idx/ss-os) ));
		arr[idx] = (1.0f - tanhf((absolute - d/2.0f)/w))/2.0f;
	}
}

cuMyMatrix Chi2Libcu::gen_kernel(unsigned int ss, unsigned int os, float d, float w){
	cuMyMatrix kernel(ss,ss);
	dim3 dimGrid(1);
	dim3 dimBlock(ss*ss);
	__gen_kernel<<<dimGrid, dimBlock>>>(kernel.devicePointer(), kernel.size(), ss, os, d, w);
	hipError_t err = hipDeviceSynchronize();
	manageError(err);

	return kernel;
}

/******************
 * Peaks
 ******************/
__device__ bool __findLocalMinimum(float* arr, unsigned int sizeX, unsigned int sizeY, unsigned int imgX, unsigned int imgY, unsigned int idx, int minsep, int* counter){
	for(int localX = minsep; localX >= -minsep; --localX){
		for(int localY = minsep; localY >= -minsep; --localY){
			if(!(localX == 0 && localY == 0)){
				int currentX = (imgX+localX);
				int currentY = (imgY+localY);

				if(currentX < 0)
					currentX = sizeX + currentX;
				if(currentY < 0)
					currentY = sizeY + currentY;

				currentX = (currentX)% sizeX;
				currentY = (currentY)% sizeY;

				if(arr[idx] <= arr[currentX+currentY*sizeY]){
					return false;
				}
			}
		}
	}
	atomicAdd(&counter[0], 1);
	return true;
}

__global__ void __findMinimums(float* arr, unsigned int sizeX, unsigned int sizeY, int threshold, int minsep, bool* out, int* counter){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int imgX = idx%sizeX;
	int imgY = (unsigned int)floorf(idx/sizeY);

	if(idx < sizeX*sizeY && arr[idx] > threshold){
		if(__findLocalMinimum(arr, sizeX, sizeY, imgX, imgY, idx, minsep, counter))
			out[idx] = true;
		else
			out[idx] = false;
	}
}

__global__ void __fillPeakArray(float* img, bool* peaks_detected, unsigned int sizeX, unsigned int sizeY, cuMyPeak* peaks, int* counter){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < sizeX*sizeY && peaks_detected[idx]){
		cuMyPeak peak;
		peak.x = (int)floorf(idx/sizeY);
		peak.y = idx%sizeX;
		peak.chi_intensity = img[idx];
		peak.fx = peak.x;
		peak.fy = peak.y;
		peak.lineal_index = idx;
		peak.img_intensity = 0;
		peak.dfx = peak.dfy = 0;
		peak.solid = false;
		peak.valid = true;
		peaks[atomicAdd(&counter[0], 1)] = peak;
	}
}

__global__ void __validatePeaks(cuMyPeak* peaks, unsigned int size, unsigned int mindistance){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int mindistance2 = mindistance*mindistance;

	if(idx < size)
	for(unsigned int j=0; j < size && j != idx ; ++j){
		int difx = peaks[idx].x - peaks[j].x;
		int dify = peaks[idx].y - peaks[j].y;

		if( (difx*difx + dify*dify) < mindistance2){
			if(peaks[idx].chi_intensity < peaks[j].chi_intensity){
				peaks[idx].valid = false;
			}else{
				peaks[j].valid = false;
			}
			break;
		}
	}
}

struct cuMyPeakCompare {
  __host__ __device__
  bool operator()(const cuMyPeak &lhs, const cuMyPeak &rhs){
	  return lhs.chi_intensity < rhs.chi_intensity;
  }
};

cuMyPeakArray Chi2Libcu::getPeaks(cuMyMatrix *arr, int threshold, int mindistance, int minsep){
	bool* d_minimums;
	size_t arrSize = arr->size()*sizeof(bool);
	hipError_t err = hipMalloc((void**)&d_minimums, arrSize);
	manageError(err);
	hipMemset(d_minimums, 0, arr->size()*sizeof(bool));

	int* h_counter; h_counter = (int*)malloc(sizeof(int));
	int* d_counter; hipMalloc((void**)&d_counter, sizeof(int));
	hipMemset(d_counter, 0, sizeof(int));

	// Encontrar Minimos
	dim3 dimGrid(_findOptimalGridSize(arr->size()));
	dim3 dimBlock(_findOptimalBlockSize(arr->size()));
	__findMinimums<<<dimGrid, dimBlock>>>(arr->devicePointer(), arr->sizeX(), arr->sizeY(), threshold, minsep, d_minimums, d_counter);
	err = hipDeviceSynchronize();
	manageError(err);

	// Contador de datos
	err = hipMemcpy(h_counter, d_counter, sizeof(int), hipMemcpyDeviceToHost);
	manageError(err);

	// Alocar datos
	cuMyPeakArray peaks(h_counter[0]);
	hipMemset(d_counter, 0, sizeof(int));

	__fillPeakArray<<<dimGrid, dimBlock>>>(arr->devicePointer(), d_minimums, arr->sizeX(), arr->sizeY(), peaks.devicePointer(), d_counter);
	err = hipDeviceSynchronize();
	manageError(err);

	// Ordenar de menor a mayor en intensidad de imagen Chi
	// TODO: Hacer un algoritmo de ordenamiento
	thrust::device_vector<cuMyPeak> dv = peaks.deviceVector();
	thrust::stable_sort(dv.begin(), dv.end(), cuMyPeakCompare());
	peaks.deviceVector(dv);

	// Validar
	dim3 dimGrid2(_findOptimalGridSize(peaks.size()));
	dim3 dimBlock2(_findOptimalBlockSize(peaks.size()));
	__validatePeaks<<<dimGrid2, dimBlock2>>>(peaks.devicePointer(), peaks.size(), mindistance);
	err = hipDeviceSynchronize();
	manageError(err);

	peaks.keepValids();

	hipFree(d_minimums); hipFree(d_counter);
	free(h_counter);
	return peaks;
}

/******************
 * Matrices Auxiliares
 ******************/

__global__ void __generateGrid(cuMyPeak* peaks, unsigned int peaks_size, unsigned int shift, float* grid_x, float* grid_y, int* over, unsigned int sizeX, unsigned int sizeY){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx >= peaks_size)
		return;

	unsigned int half=(shift+2);
	int currentX, currentY;
	float currentDistance = 0.0;
	float currentDistanceAux = 0.0;

	if(peaks_size != 0){
		for(unsigned int localX=0; localX < 2*half+1; ++localX){
			for(unsigned int localY=0; localY < 2*half+1; ++localY){
				cuMyPeak currentPeak = peaks[idx];
				currentX = (int)round(currentPeak.fx) - shift + (localX - half);
				currentY = (int)round(currentPeak.fy) - shift + (localY - half);

				if( 0 <= currentX && currentX < sizeX && 0 <= currentY && currentY < sizeY ){
					int index = currentX+sizeY*currentY;
					currentDistance =
							sqrtf(grid_x[index]*grid_x[index] + grid_y[index]*grid_y[index]);

					currentDistanceAux =
							sqrtf(1.0f*(1.0f*localX-half+currentPeak.x - currentPeak.fx)*(1.0f*localX-half+currentPeak.x - currentPeak.fx) +
								  1.0f*(1.0f*localY-half+currentPeak.y - currentPeak.fy)*(1.0f*localY-half+currentPeak.y - currentPeak.fy));

					if(currentDistance >= currentDistanceAux){
						over[index] = idx+1;
						grid_x[index] = (1.0f*localX-half+currentPeak.x)-currentPeak.fx;
						grid_y[index] = (1.0f*localY-half+currentPeak.y)-currentPeak.fy;
					}
				}
			}
		}
	}
}

void Chi2Libcu::generateGrid(cuMyPeakArray* peaks, unsigned int shift, cuMyMatrix* grid_x, cuMyMatrix* grid_y, cuMyMatrixi* over){
	unsigned int maxDimension = grid_x->sizeX() > grid_x->sizeY() ? grid_x->sizeX() : grid_x->sizeY();
	grid_x->reset(maxDimension);
	grid_y->reset(maxDimension);
	over->reset(0);

	dim3 dimGrid(_findOptimalGridSize(peaks->size()));
	dim3 dimBlock(_findOptimalBlockSize(peaks->size()));
	__generateGrid<<<dimGrid, dimBlock>>>(peaks->devicePointer(), peaks->size(), shift, grid_x->devicePointer(), grid_y->devicePointer(), over->devicePointer(), grid_x->sizeX(), grid_x->sizeY());
	hipError_t err = hipDeviceSynchronize();
	manageError(err);
}

/******************
 * Chi2 Difference
 ******************/
__global__ void __computeDifference(float* img, float* grid_x, float* grid_y, float d, float w, float* diffout, unsigned int size){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx > size)
		return;

	float x2y2 = sqrtf(1.0f*grid_x[idx]*grid_x[idx] + 1.0f*grid_y[idx]*grid_y[idx]);
	float temp = ((1.0f-tanhf( (x2y2-d/2.0)/w )) - 2.0f*img[idx])/2.0f;

	diffout[idx] = temp;
}

int Chi2Libcu::computeDifference(cuMyMatrix *img, cuMyMatrix *grid_x, cuMyMatrix *grid_y, float d, float w, cuMyMatrix *diffout){
	dim3 dimGrid(_findOptimalGridSize(img->size()));
	dim3 dimBlock(_findOptimalBlockSize(img->size()));
	__computeDifference<<<dimGrid, dimBlock>>>(img->devicePointer(), grid_x->devicePointer(), grid_y->devicePointer(), d, w, diffout->devicePointer(), img->size());
	hipError_t err = hipDeviceSynchronize();
	manageError(err);

	// TODO: Sumar lo calculado para obtener el error Chi2

	return 0;
}
