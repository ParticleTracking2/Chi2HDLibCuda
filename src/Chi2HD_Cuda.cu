#include "hip/hip_runtime.h"
//============================================================================
// Name        : Chi2HD_Cuda.cu
// Author      : Juan Silva
// Version     :
// Copyright   : All rights reserved
// Description : Funciones para trabajar el algoritmo de minimos cuadrados
//============================================================================

#include "Headers/Chi2HD_Cuda.h"
#include "Headers/Chi2HD_CudaUtils.h"

/**
 * Elementos a considerar con 275 GTX
 * Global Memory = 896 MB
 * Const Memory = 65KB (16000 Floats)
 * Shared Memory = 16KB (Compartida en el bloque = 4000 Floats)
 * Registros por Bloque = 16K
 * ------------------------------
 * Cuda Cores = 240 (30 Multiprosessors)* (8 CUDA Cores/MP)
 * Max Threads x Bloque = 512
 * Maximas dimensiones por bloque = 512 x 512 x 64
 * Maximas dimensiones de una grilla = 65535 x 65535 x 1
 */

#if defined(__cplusplus)
extern "C" {
#endif

/******************
 * Creacion y destruccion de arreglos
 ******************/
cuMyArray2D CHI2HD_createArray(unsigned int sx, unsigned int sy){
	cuMyArray2D ret;
	hipError_t err = hipMalloc((void**)&ret._device_array, (size_t)(sy*sx*sizeof(float)));
	manageError(err);

	ret._host_array = 0;
	ret._device = 0;
	ret._sizeX = sx;
	ret._sizeY = sy;

	return ret;
}

void CHI2HD_createArrayPointer(unsigned int sx, unsigned int sy, cuMyArray2D* ret){
	hipError_t err = hipMalloc((void**)&ret->_device_array, (size_t)(sy*sx*sizeof(float)));
	manageError(err);

	ret->_host_array = 0;
	ret->_device = 0;
	ret->_sizeX = sx;
	ret->_sizeY = sy;
}

bool CHI2HD_destroyArray(cuMyArray2D *arr){
	if(arr->_device_array){
		hipError_t err = hipFree(arr->_device_array);
		manageError(err);
	}
	if(arr->_host_array)
		free(arr->_host_array);

	arr->_device_array = 0;
	arr->_host_array = 0;
	arr->_device = -1;
	arr->_sizeX = 0;
	arr->_sizeY = 0;
	return true;
}

/******************
 * RESET
 ******************/
__global__ void __CHI2HD_reset(float* arr, int size, float def){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
		arr[idx] = def;
}

void CHI2HD_reset(cuMyArray2D *arr, float def){
	dim3 dimGrid(_findOptimalGridSize(arr));
	dim3 dimBlock(_findOptimalBlockSize(arr));
	__CHI2HD_reset<<<dimGrid, dimBlock>>>(arr->_device_array, arr->getSize(), def);
	hipError_t err = hipDeviceSynchronize();
	manageError(err);
}

/******************
 * SQUARE
 ******************/
__global__ void __CHI2HD_squareIt(float* arr, unsigned int size){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
		arr[idx] = arr[idx] * arr[idx];
}

void CHI2HD_squareIt(cuMyArray2D *arr){
	dim3 dimGrid(_findOptimalGridSize(arr));
	dim3 dimBlock(_findOptimalBlockSize(arr));
	__CHI2HD_squareIt<<<dimGrid, dimBlock>>>(arr->_device_array, arr->getSize());
	hipError_t err = hipDeviceSynchronize();
	manageError(err);
}

/******************
 * CUBE
 ******************/
__global__ void __CHI2HD_cubeIt(float* arr, int size){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
		arr[idx] = arr[idx] * arr[idx] * arr[idx];
}

void CHI2HD_cubeIt(cuMyArray2D *arr){
	dim3 dimGrid(_findOptimalGridSize(arr));
	dim3 dimBlock(_findOptimalBlockSize(arr));
	__CHI2HD_cubeIt<<<dimGrid, dimBlock>>>(arr->_device_array, arr->getSize());
	hipError_t err = hipDeviceSynchronize();
	manageError(err);
}

/******************
 * Copy
 ******************/
void CHI2HD_copy(cuMyArray2D *src, cuMyArray2D *dst){
	if(!dst->_device_array){
		 cuMyArray2D tmp = CHI2HD_createArray(src->_sizeX, src->_sizeY);
		 dst->_device_array = tmp._device_array;
		 dst->_sizeX = tmp._sizeX;
		 dst->_sizeY = tmp._sizeY;
	}
	hipError_t err = hipMemcpy(dst->_device_array, src->_device_array, src->getSize()*sizeof(float), hipMemcpyDeviceToDevice);
	manageError(err);
}

void CHI2HD_copyToHost(cuMyArray2D *arr){
	size_t size = arr->getSize()*sizeof(float);
	if(!arr->_host_array)
		arr->_host_array = (float*)malloc(size);
	hipError_t err = hipMemcpy(arr->_host_array, arr->_device_array, size, hipMemcpyDeviceToHost);
	manageError(err);
}

void CHI2HD_copyToDevice(cuMyArray2D *arr){
	size_t size = arr->getSize()*sizeof(float);
	hipError_t err;
	if(!arr->_device_array){
		err = hipMalloc((void**)&arr->_device_array, arr->_sizeX*sizeof(float)*arr->_sizeY);
		manageError(err);
	}
	err = hipMemcpy(arr->_device_array, arr->_host_array, size, hipMemcpyHostToDevice);
	manageError(err);
}

/******************
 * Min Max
 ******************/
myPair CHI2HD_minMax(cuMyArray2D *arr){
	myPair ret;
	if(!arr->_host_array)
		CHI2HD_copyToHost(arr);

	float tempMax = arr->getValueHost(0);
	float tempMin = arr->getValueHost(0);
	for(unsigned int i=0; i < arr->getSize(); ++i){
		float tmp = arr->_host_array[i];
		if(tempMax < tmp)
			tempMax = tmp;
		if(tempMin > tmp)
			tempMin = tmp;
	}
	ret.first = tempMin;
	ret.second = tempMax;

	return ret;
}

/******************
 * Normalizar
 ******************/
__global__ void __CHI2HD_normalize(float* arr, unsigned int size, float _min, float _max){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float dif = _max - _min;
	if(idx < size)
		arr[idx] = (float)((1.0f*_max - arr[idx]*1.0f)/dif);
}

void CHI2HD_normalize(cuMyArray2D *arr, float _min, float _max){
	dim3 dimGrid(_findOptimalGridSize(arr));
	dim3 dimBlock(_findOptimalBlockSize(arr));
	__CHI2HD_normalize<<<dimGrid, dimBlock>>>(arr->_device_array, arr->getSize(), _min, _max);
	hipError_t err = hipDeviceSynchronize();
	manageError(err);
}

/******************
 * Kernel
 ******************/
__global__ void __CHI2HD_gen_kernel(float* arr, unsigned int size, unsigned int ss, unsigned int os, float d, float w){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size){
		float absolute = abs(sqrtf( (idx%ss-os)*(idx%ss-os) + (idx/ss-os)*(idx/ss-os) ));
		arr[idx] = (1.0f - tanhf((absolute - d/2.0f)/w))/2.0f;
	}
}

cuMyArray2D CHI2HD_gen_kernel(unsigned int ss, unsigned int os, float d, float w){
	cuMyArray2D kernel = CHI2HD_createArray(ss,ss);
	dim3 dimGrid(1);
	dim3 dimBlock(ss*ss);
	__CHI2HD_gen_kernel<<<dimGrid, dimBlock>>>(kernel._device_array, kernel.getSize(), ss, os, d, w);
	hipError_t err = hipDeviceSynchronize();
	manageError(err);
	return kernel;
}

#if defined(__cplusplus)
}
#endif
