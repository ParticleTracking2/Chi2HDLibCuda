#include "hip/hip_runtime.h"
//============================================================================
// Name        : Chi2HD_CudaFFT.cu
// Author      : Juan Silva
// Version     :
// Copyright   : All rights reserved
// Description : Funciones para trabajar el algoritmo de minimos cuadrados. Parte FFT
//============================================================================


#include "Headers/Chi2HD_CudaFFT.h"
#include "Headers/Chi2HD_CudaUtils.h"

#if defined(__cplusplus)
extern "C" {
#endif

/**
 * Maneja los errores de CUDA
 */
void manageErrorFFT(hipfftResult res){
	if(res != HIPFFT_SUCCESS){
		printf("CHI2HD_CUDA FFT Error: ");
		switch (res) {
			case HIPFFT_INVALID_PLAN:
				printf("Plan Invalido\n");
				break;
			case HIPFFT_INVALID_TYPE:
				printf("Tipo invalido\n");
				break;
			case HIPFFT_INVALID_VALUE:
				printf("Valor invalido\n");
				break;
			case HIPFFT_INTERNAL_ERROR:
				printf("Error inerno\n");
				break;
			case HIPFFT_EXEC_FAILED:
				printf("Falla de ejecucion\n");
				break;
			case HIPFFT_SETUP_FAILED:
				printf("Falla de setup\n");
				break;
			case HIPFFT_INVALID_SIZE:
				printf("Tamaño invalido\n");
				break;
			case HIPFFT_UNALIGNED_DATA:
				printf("Datos desalineados\n");
				break;
			default:
				printf("Error Desconocido\n");
				break;
		}
		exit(-1);
	}
}

/******************
 * Modula y Normaliza cada elemento de la transformacion.
 * Guarda los resultados en img.
 ******************/
__global__ void __CHI2HD_modulateAndNormalize(hipfftComplex* img, hipfftComplex* kernel, float nwnh, int limit){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < limit){
		float f1 = img[idx].x*kernel[idx].x - img[idx].y*kernel[idx].y;
		float f2 = img[idx].x*kernel[idx].y + img[idx].y*kernel[idx].x;

		img[idx].x=f1*nwnh;
		img[idx].y=f2*nwnh;
	}
}

/******************
 * Copia la matriz transpuesta
 ******************/
__global__ void __CHI2HD_copyInside(hipfftReal* container, unsigned int container_sizeX, unsigned int container_sizeY, float* data, unsigned int data_sizeX, unsigned int data_sizeY){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	int add = floorf(idx/data_sizeX)*(container_sizeX-data_sizeX); // Normal
	int add = floorf(idx%data_sizeX)*container_sizeX-idx + floorf(idx/data_sizeX); // Transpuesta
	if(idx < data_sizeX*data_sizeY){
		container[idx+add] = data[idx];
	}
}

/**
 * Para efectos de Debug
 * Escribe un archivo con los datos de hipfftReal
 */
void writeDataR(hipfftReal* data, int width, int height, char* filestr){
	FILE *file;
	file = fopen(filestr,"w+");
	for(int x=0; x < width; ++x){
		for(int y=0; y < height; ++y){
			fprintf(file,"%f;", data[x+y*height]);
		}
		fprintf(file,"\n");
	}
	fclose(file);
}

/**
 * Para efectos de Debug
 * Escribe un archivo con los datos de hipfftComplex en forma lineal
 */
void writeDataC(hipfftComplex* data, int size, char* filestr){
	FILE *file;
	file = fopen(filestr,"w+");
	for(int x=0; x < size; ++x){
		fprintf(file,"%fR;%fI;\n", data[x].x, data[x].y);
	}
	fclose(file);
}

/******************
 * Convolucion 2D
 * Usando Zero Padding
 ******************/
void CHI2HD_conv2D(cuMyArray2D* img, cuMyArray2D* kernel_img, cuMyArray2D* output){
	hipfftHandle plan_forward_image, plan_forward_kernel, plan_backward;
	hipfftComplex *fft_image, *fft_kernel;
	hipfftReal *ifft_result, *data, *kernel; // float *

	int nwidth 	=	output->_sizeX; //(int)(img->_sizeX+kernel_img->_sizeX-1);
	int nheight	=	output->_sizeY; //(int)(img->_sizeY+kernel_img->_sizeY-1);
	// Input Complex Data
	hipError_t err;
	err = hipMalloc((void**)&fft_image, sizeof(hipfftComplex)*(nwidth*(nheight/2 +1)));
	manageError(err);
	err = hipMalloc((void**)&fft_kernel, sizeof(hipfftComplex)*(nwidth*(nheight/2 +1)));
	manageError(err);
	// Output Real Data
	err = hipMalloc((void**)&ifft_result, sizeof(hipfftReal)*nwidth*nheight);
	manageError(err);
	err = hipMalloc((void**)&data, sizeof(hipfftReal)*nwidth*nheight);
	manageError(err);
	err = hipMalloc((void**)&kernel, sizeof(hipfftReal)*nwidth*nheight);
	manageError(err);

	// Plans
	hipfftResult res = hipfftPlan2d(&plan_forward_image, nwidth, nheight, HIPFFT_R2C);
	manageErrorFFT(res);
	res = hipfftPlan2d(&plan_forward_kernel, nwidth, nheight, HIPFFT_R2C);
	manageErrorFFT(res);
	res = hipfftPlan2d(&plan_backward, nwidth, nheight, HIPFFT_C2R);
	manageErrorFFT(res);

	// Populate Data
	err = hipMemset((void*)data, 0, nwidth*nheight*sizeof(hipfftReal));
	manageError(err);
	err = hipMemset((void*)kernel, 0, nwidth*nheight*sizeof(hipfftReal));
	manageError(err);

	dim3 dimGrid0(_findOptimalGridSize(img));
	dim3 dimBlock0(_findOptimalBlockSize(img));
	__CHI2HD_copyInside<<<dimGrid0, dimBlock0>>>(data, nwidth, nheight, img->_device_array, img->_sizeX, img->_sizeY);
	err = hipDeviceSynchronize();
	manageError(err);

	dim3 dimGrid1(_findOptimalGridSize(kernel_img));
	dim3 dimBlock1(_findOptimalBlockSize(kernel_img));
	__CHI2HD_copyInside<<<dimGrid1, dimBlock1>>>(kernel, nwidth, nheight, kernel_img->_device_array, kernel_img->_sizeX, kernel_img->_sizeY);
	err = hipDeviceSynchronize();
	manageError(err);

	/* FFT Execute */
		// Execute Plan
		res = hipfftExecR2C(plan_forward_image, data, fft_image);
		manageErrorFFT(res);
		err = hipDeviceSynchronize();
		manageError(err);

		res = hipfftExecR2C(plan_forward_kernel, kernel, fft_kernel);
		manageErrorFFT(res);
		err = hipDeviceSynchronize();
		manageError(err);

		// Modular y Normalizar
		dim3 dimGrid2(_findOptimalGridSize(output));
		dim3 dimBlock2(_findOptimalBlockSize(output));
		__CHI2HD_modulateAndNormalize<<<dimGrid2, dimBlock2>>>(fft_image, fft_kernel, (float)(1.0f/(float)(nwidth*nheight)), (int)(nwidth *(nheight/2 +1)));
		err = hipDeviceSynchronize();
		manageError(err);

		// Execute Plan
		res = hipfftExecC2R(plan_backward, fft_image, ifft_result);
		manageErrorFFT(res);
		err = hipDeviceSynchronize();
		manageError(err);
	/* FFT Execute */

	// Copy Result to output;
	err = hipMemcpy(output->_device_array, ifft_result, sizeof(hipfftReal)*nwidth*nheight, hipMemcpyDeviceToDevice);
	manageError(err);

	hipfftDestroy(plan_forward_image);
	hipfftDestroy(plan_forward_kernel);
	hipfftDestroy(plan_backward);
	err = hipFree(data); manageError(err);
	err = hipFree(kernel); manageError(err);
	err = hipFree(ifft_result); manageError(err);
	err = hipFree(fft_image); manageError(err);
	err = hipFree(fft_kernel); manageError(err);
}

/******************
 * Calcula el resultado general de las transformaciones
 ******************/
__global__ void __CHI2HD_fftresutl(float* first_term, float* second_term, float* third_term, float* output, unsigned int size){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size){
		output[idx] = 1.0f/(1.0f + (-2.0f*first_term[idx] + second_term[idx])/third_term[idx]);
	}
}

void CHI2HD_fftresutl(cuMyArray2D* first_term, cuMyArray2D* second_term, cuMyArray2D* third_term, cuMyArray2D* output){
	dim3 dimGrid(_findOptimalGridSize(output));
	dim3 dimBlock(_findOptimalBlockSize(output));
	__CHI2HD_fftresutl<<<dimGrid, dimBlock>>>(first_term->_device_array, second_term->_device_array, third_term->_device_array, output->_device_array, output->getSize());
	hipError_t err = hipDeviceSynchronize();
	manageError(err);
}


#if defined(__cplusplus)
}
#endif
