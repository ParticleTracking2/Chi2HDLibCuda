#include "hip/hip_runtime.h"
//============================================================================
// Name        : Chi2HD_CudaFFT.cu
// Author      : Juan Silva
// Version     :
// Copyright   : All rights reserved
// Description : Funciones para trabajar el algoritmo de minimos cuadrados. Parte FFT
//============================================================================


#include "Headers/Chi2HD_CudaUtils.h"

#if defined(__cplusplus)
extern "C" {
#endif

/******************
 * Utilidades
 ******************/
/**
 * Numeros de bloques a ejecutar.
 */
unsigned int _findOptimalGridSize(cuMyArray2D *arr){
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	unsigned int maxThreads = deviceProp.maxThreadsPerBlock;
	unsigned int requiredGrid = ceil(arr->getSize()/maxThreads)+1;
	if(requiredGrid < deviceProp.maxGridSize[0])
		return requiredGrid;
	return deviceProp.maxGridSize[0];
}

/**
 * Cantidad de threads dentro de cada bloque.
 */
unsigned int _findOptimalBlockSize(cuMyArray2D *arr){
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	return deviceProp.maxThreadsPerBlock;
}

/**
 * Maneja los errores de CUDA
 */
void manageError(hipError_t err){
	if(err != hipSuccess){
		printf("CHI2HD_CUDA Error: %s\n", hipGetErrorString(err));
		exit(-1);
	}
}

#if defined(__cplusplus)
}
#endif
