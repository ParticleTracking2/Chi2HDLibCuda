#include "hip/hip_runtime.h"
/*
 * Chi2LibcuFFT.cu
 *
 *  Created on: 10/12/2011
 *      Author: juanin
 */

#include "Headers/Chi2LibcuFFT.h"
#include "Headers/Chi2LibcuUtils.h"

/**
 * Maneja los errores de CUDA
 */
void Chi2LibcuFFT::manageErrorFFT(hipfftResult res){
	if(res != HIPFFT_SUCCESS){
		printf("CHI2HD_CUDA FFT Error: ");
		switch (res) {
			case HIPFFT_INVALID_PLAN:
				printf("Plan Invalido\n");
				break;
			case HIPFFT_INVALID_TYPE:
				printf("Tipo invalido\n");
				break;
			case HIPFFT_INVALID_VALUE:
				printf("Valor invalido\n");
				break;
			case HIPFFT_INTERNAL_ERROR:
				printf("Error inerno\n");
				break;
			case HIPFFT_EXEC_FAILED:
				printf("Falla de ejecucion\n");
				break;
			case HIPFFT_SETUP_FAILED:
				printf("Falla de setup\n");
				break;
			case HIPFFT_INVALID_SIZE:
				printf("Tamaño invalido\n");
				break;
			case HIPFFT_UNALIGNED_DATA:
				printf("Datos desalineados\n");
				break;
			default:
				printf("Error Desconocido\n");
				break;
		}
		exit(-1);
	}
}

/******************
 * Modula y Normaliza cada elemento de la transformacion.
 * Guarda los resultados en img.
 ******************/
__global__ void __Chi2LibcuFFT_modulateAndNormalize(hipfftComplex* img, hipfftComplex* kernel, float nwnh, int limit){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < limit){
		float f1 = img[idx].x*kernel[idx].x - img[idx].y*kernel[idx].y;
		float f2 = img[idx].x*kernel[idx].y + img[idx].y*kernel[idx].x;

		img[idx].x=f1*nwnh;
		img[idx].y=f2*nwnh;
	}
}

/******************
 * Copia la matriz transpuesta
 ******************/
__global__ void __Chi2LibcuFFT_copyInside(hipfftReal* container, unsigned int container_sizeX, unsigned int container_sizeY, float* data, unsigned int data_sizeX, unsigned int data_sizeY){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	int add = floorf(idx/data_sizeX)*(container_sizeX-data_sizeX); // Normal
	int add = floorf(idx%data_sizeX)*container_sizeX-idx + floorf(idx/data_sizeX); // Transpuesta
	if(idx < data_sizeX*data_sizeY){
		container[idx+add] = data[idx];
	}
}

/******************
 * Convolucion 2D
 * Usando Zero Padding
 ******************/
void Chi2LibcuFFT::conv2D(cuMyMatrix* img, cuMyMatrix* kernel_img, cuMyMatrix* output){
	hipfftHandle plan_forward_image, plan_forward_kernel, plan_backward;
	hipfftComplex *fft_image, *fft_kernel;
	hipfftReal *ifft_result, *data, *kernel; // float *

	int nwidth 	=	output->sizeX(); //(int)(img->_sizeX+kernel_img->_sizeX-1);
	int nheight	=	output->sizeY(); //(int)(img->_sizeY+kernel_img->_sizeY-1);
	// Input Complex Data
	hipError_t err;
	err = hipMalloc((void**)&fft_image, sizeof(hipfftComplex)*(nwidth*(nheight/2 +1)));
	manageError(err);
	err = hipMalloc((void**)&fft_kernel, sizeof(hipfftComplex)*(nwidth*(nheight/2 +1)));
	manageError(err);
	// Output Real Data
	err = hipMalloc((void**)&ifft_result, sizeof(hipfftReal)*nwidth*nheight);
	manageError(err);
	err = hipMalloc((void**)&data, sizeof(hipfftReal)*nwidth*nheight);
	manageError(err);
	err = hipMalloc((void**)&kernel, sizeof(hipfftReal)*nwidth*nheight);
	manageError(err);

	// Plans
	hipfftResult res = hipfftPlan2d(&plan_forward_image, nwidth, nheight, HIPFFT_R2C);
	manageErrorFFT(res);
	res = hipfftPlan2d(&plan_forward_kernel, nwidth, nheight, HIPFFT_R2C);
	manageErrorFFT(res);
	res = hipfftPlan2d(&plan_backward, nwidth, nheight, HIPFFT_C2R);
	manageErrorFFT(res);

	// Populate Data
	err = hipMemset((void*)data, 0, nwidth*nheight*sizeof(hipfftReal));
	manageError(err);
	err = hipMemset((void*)kernel, 0, nwidth*nheight*sizeof(hipfftReal));
	manageError(err);

	dim3 dimGrid0(_findOptimalGridSize(img->size()));
	dim3 dimBlock0(_findOptimalBlockSize(img->size()));
	__Chi2LibcuFFT_copyInside<<<dimGrid0, dimBlock0>>>(data, nwidth, nheight, img->devicePointer(), img->sizeX(), img->sizeY());
	checkAndSync();

	dim3 dimGrid1(_findOptimalGridSize(kernel_img->size()));
	dim3 dimBlock1(_findOptimalBlockSize(kernel_img->size()));
	__Chi2LibcuFFT_copyInside<<<dimGrid1, dimBlock1>>>(kernel, nwidth, nheight, kernel_img->devicePointer(), kernel_img->sizeX(), kernel_img->sizeY());
	checkAndSync();

	/* FFT Execute */
		// Execute Plan
		res = hipfftExecR2C(plan_forward_image, data, fft_image);
		manageErrorFFT(res);
		err = hipDeviceSynchronize();
		manageError(err);

		res = hipfftExecR2C(plan_forward_kernel, kernel, fft_kernel);
		manageErrorFFT(res);
		err = hipDeviceSynchronize();
		manageError(err);

		// Modular y Normalizar
		dim3 dimGrid2(_findOptimalGridSize(output->size()));
		dim3 dimBlock2(_findOptimalBlockSize(output->size()));
		__Chi2LibcuFFT_modulateAndNormalize<<<dimGrid2, dimBlock2>>>(fft_image, fft_kernel, (float)(1.0f/(float)(nwidth*nheight)), (int)(nwidth *(nheight/2 +1)));
		checkAndSync();

		// Execute Plan
		res = hipfftExecC2R(plan_backward, fft_image, ifft_result);
		manageErrorFFT(res);
		err = hipDeviceSynchronize();
		manageError(err);
	/* FFT Execute */

	// Copy Result to output;
	err = hipMemcpy(output->devicePointer(), ifft_result, sizeof(hipfftReal)*nwidth*nheight, hipMemcpyDeviceToDevice);
	manageError(err);

	hipfftDestroy(plan_forward_image);
	hipfftDestroy(plan_forward_kernel);
	hipfftDestroy(plan_backward);
	err = hipFree(data); manageError(err);
	err = hipFree(kernel); manageError(err);
	err = hipFree(ifft_result); manageError(err);
	err = hipFree(fft_image); manageError(err);
	err = hipFree(fft_kernel); manageError(err);
}

/******************
 * Calcula el resultado general de las transformaciones
 ******************/
__global__ void __Chi2LibcuFFT_fftresutl(float* first_term, float* second_term, float* third_term, float* output, unsigned int size){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size){
		output[idx] = 1.0f/(1.0f + (-2.0f*first_term[idx] + second_term[idx])/third_term[idx]);
	}
}

void Chi2LibcuFFT::fftresutl(cuMyMatrix* first_term, cuMyMatrix* second_term, cuMyMatrix* third_term, cuMyMatrix* output){
	dim3 dimGrid(_findOptimalGridSize(output->size()));
	dim3 dimBlock(_findOptimalBlockSize(output->size()));
	__Chi2LibcuFFT_fftresutl<<<dimGrid, dimBlock>>>(first_term->devicePointer(), second_term->devicePointer(), third_term->devicePointer(), output->devicePointer(), output->size());
	checkAndSync();
}


