/*
 * cuMyPeakArray.cu
 *
 *  Created on: 11/12/2011
 *      Author: juanin
 */

#include "../Headers/Container/cuMyPeak.h"
#include "../Headers/Chi2LibcuUtils.h"
#include <iostream>

void cuMyPeakArray::goEmpty(){
	_host_array = 0;
	_device_array = 0;
	_size = 0;
}

/**
 *******************************
 * Constructores y Destructores
 *******************************
 */
cuMyPeakArray::cuMyPeakArray(){
	goEmpty();
}

cuMyPeakArray::cuMyPeakArray(unsigned int size){
	goEmpty();
	_size = size;
	allocateDevice();
}

void cuMyPeakArray::allocateDevice(){
	if(_size > 0){
		hipError_t err = hipMalloc((void**)&_device_array, (size_t)(_size*sizeof(cuMyPeak)));
		manageError(err);
	}
}

void cuMyPeakArray::allocateHost(){
	if(_size > 0)
		_host_array = (cuMyPeak*)malloc(_size*sizeof(cuMyPeak));
}

cuMyPeakArray::~cuMyPeakArray(){
	deallocateDevice();
}

void cuMyPeakArray::deallocateDevice(){
	if(_device_array){
		hipError_t err = hipFree(_device_array);
		manageError(err);
	}
	_device_array = 0;
	if(!_host_array)
		_size = 0;
}

void cuMyPeakArray::deallocateHost(){
	if(_host_array){
		free(_host_array);
	}
	_host_array = 0;
	if(!_device_array)
		_size = 0;
}

/**
 *******************************
 * Metodos
 *******************************
 */

void cuMyPeakArray::copyToHost(){
	if(!_host_array){
		allocateHost();
	}
	hipError_t err = hipMemcpy(_host_array, _device_array, _size*sizeof(cuMyPeak), hipMemcpyDeviceToHost);
	manageError(err);
}

void cuMyPeakArray::copyToDevice(){
	if(!_device_array){
		allocateDevice();
	}
	hipError_t err = hipMemcpy(_device_array, _host_array, _size*sizeof(cuMyPeak), hipMemcpyHostToDevice);
	manageError(err);
}

unsigned int cuMyPeakArray::size(){
	return _size;
}

cuMyPeak* cuMyPeakArray::devicePointer(){
	return _device_array;
}

cuMyPeak* cuMyPeakArray::hostPointer(){
	return _host_array;
}

void cuMyPeakArray::keepValids(){
	//Contar Validos
	copyToHost();
	cuMyPeak tmp[_size];

	unsigned int valids = 0;
	for(unsigned int i=0; i < _size; ++i){
		if(_host_array[i].valid){
			tmp[valids] = _host_array[i];
			++valids;
		}
	}

	// Borrar datos y copiar
	deallocateDevice();	deallocateHost();
	_size = valids;
	allocateHost();
	for(unsigned int i=0; i < _size; ++i){
		_host_array[i] = tmp[i];
	}
	copyToDevice();
}

thrust::device_vector<cuMyPeak> cuMyPeakArray::deviceVector(){
	thrust::host_vector<cuMyPeak> hv(_size);
	copyToHost();
	for(unsigned int i=0; i < _size; ++i){
		hv[i] = _host_array[i];
	}
	thrust::device_vector<cuMyPeak> ret = hv;
	return ret;
}
void cuMyPeakArray::deviceVector(thrust::device_vector<cuMyPeak> dv){
	thrust::host_vector<cuMyPeak> hv = dv;
	for(unsigned int i=0; i < _size; ++i){
		_host_array[i] = hv[i];
	}
	copyToDevice();
}

cuMyPeak cuMyPeakArray::getHostValue(unsigned int index){
	return _host_array[index];
}

cuMyPeak & cuMyPeakArray::atHost(unsigned int index){
	return _host_array[index];
}
